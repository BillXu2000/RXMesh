#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include "gtest/gtest.h"
#include "rxmesh/rxmesh_attribute.h"
#include "rxmesh/rxmesh_static.h"
#include "rxmesh/util/import_obj.h"
#include "rxmesh/util/report.h"
#include "rxmesh/util/timer.h"
#include "mass_spring_kernel.cuh"
#include <iostream>

struct arg
{
    std::string obj_file_name = STRINGIFY(INPUT_DIR) "sphere3.obj";
    std::string output_folder = STRINGIFY(OUTPUT_DIR);
    uint32_t    num_run = 1;
    uint32_t    device_id = 0;
    char**      argv;
    int         argc;
    bool        shuffle = false;
    bool        sort = false;
    bool ev = false;
    bool vv = false;
} Arg;

template <typename T, uint32_t patchSize>
void mass_spring_rxmesh(RXMESH::RXMeshStatic<patchSize>&          rxmesh_static,
                        const std::vector<std::vector<T>>&        Verts,
                        const std::vector<std::vector<uint32_t>>& Faces)
{
    using namespace RXMESH;
    constexpr uint32_t blockThreads = 256;

    // Report
    Report report("MassSpring_RXMesh");
    report.command_line(Arg.argc, Arg.argv);
    report.device();
    report.system();
    report.model_data(Arg.obj_file_name, rxmesh_static);
    report.add_member("method", std::string("RXMesh"));
    std::string order = "default";
    if (Arg.shuffle) {
        order = "shuffle";
    } else if (Arg.sort) {
        order = "sorted";
    }
    report.add_member("input_order", order);
    report.add_member("blockThreads", blockThreads);

    RXMeshAttribute<T> ox, x;
    ox.set_name("ox");
	x.set_name("x");
    ox.init(Verts.size(), 3u, RXMESH::LOCATION_ALL);
	x.init(Verts.size(), 3u, RXMESH::LOCATION_ALL);
    // fill in the coordinates
    float mi[3], ma[3];
    for (uint32_t i = 0; i < Verts.size(); ++i) {
        for (uint32_t j = 0; j < Verts[i].size(); ++j) {
            if (i == 0) {
                mi[j] = ma[j] = Verts[i][j];
            }
            mi[j] = min(mi[j], Verts[i][j]);
            ma[j] = max(ma[j], Verts[i][j]);
        }
    }
    for (uint32_t i = 0; i < Verts.size(); ++i) {
        for (uint32_t j = 0; j < Verts[i].size(); ++j) {
            ox(i, j) = x(i, j) = (Verts[i][j] - mi[j]) / (ma[1] - mi[1]);
        }
    }
    auto print_momentum = [&]() {
        float sum[4];
        memset(sum, 0, sizeof(sum));
        for (int i = 0; i < Verts.size(); i++) {
            for (int j = 0; j < Verts[i].size(); j++) {
                float product = 1;
                for (int k = 0; k < 4; k++) {
                    sum[k] += product;
                    product *= x(i, j);
                }
            }
        }
        std::cerr << "momentum:\t";
        for (int i = 0; i < 4; i++) {
            std::cerr << sum[i] << "\t";
        }
        std::cerr << "\n";
    };
    auto print_obj = [&](std::string path) {
        std::fstream out(path, std::fstream::out);
        for (int i = 0; i < Verts.size(); i++) {
            out << "v " << x(i, 0) << " " << x(i, 1) << " " << x(i, 2) << "\n";
        }
        for (auto f : Faces) {
            out << "f " << f[0] + 1 << " " << f[1] + 1 << " " << f[2] + 1 << "\n";
        }
    };
    print_momentum();
    // move the coordinates to device
    ox.move(RXMESH::HOST, RXMESH::DEVICE);
		x.move(RXMESH::HOST, RXMESH::DEVICE);

    // velocity
    RXMeshAttribute<T> v;
    v.set_name("v");
    v.init(Verts.size(), 3u, RXMESH::LOCATION_ALL);

    // launch box
    LaunchBox<blockThreads> launch_box;
    rxmesh_static.prepare_launch_box(RXMESH::Op::VV, launch_box);

    LaunchBox<blockThreads> ev_launch_box;
    rxmesh_static.prepare_launch_box(RXMESH::Op::EV, ev_launch_box);


    TestData td;
    td.test_name = "MassSpring";

    float mass = 1.0 / Verts.size();
    std::cerr << mass << "\n";
    for (uint32_t itr = 0; itr < Arg.num_run; ++itr) {
        for (int j = 0; j < 100; j++) {
            if (Arg.vv)
                compute_mass_spring<T, blockThreads>
                    <<<launch_box.blocks, blockThreads, launch_box.smem_bytes_dyn>>>(
                        rxmesh_static.get_context(), ox, x, v, mass);
            if (Arg.ev) 
                ev_mass_spring<T, blockThreads>
                    <<<ev_launch_box.blocks, blockThreads, ev_launch_box.smem_bytes_dyn>>>(
                        rxmesh_static.get_context(), ox, x, v, mass);
            advect<T, blockThreads>
                <<<launch_box.blocks, blockThreads, launch_box.smem_bytes_dyn>>>(
                    rxmesh_static.get_context(), ox, x, v, mass);
        }
            
        /*advance<T, blockThreads>
            <<<launch_box.blocks, blockThreads, launch_box.smem_bytes_dyn>>>(
                rxmesh_static.get_context(), ox, x, v);*/
        /*x.move(RXMESH::DEVICE, RXMESH::HOST);
        print_momentum();
        print_obj("./results/" + std::to_string(itr) + ".obj");*/
    }
    x.move(RXMESH::DEVICE, RXMESH::HOST);
    print_momentum();


 
    // Release allocation
    ox.release();
    x.release();
		v.release();

    // Finalize report
    report.add_test(td);
    report.write(Arg.output_folder + "/rxmesh",
                 "MassSpring_RXMesh_" + extract_file_name(Arg.obj_file_name));
}

TEST(Apps, MassSpring)
{
    using namespace RXMESH;
    using dataT = float;

    if (Arg.shuffle) {
        ASSERT_FALSE(Arg.sort) << " cannot shuffle and sort at the same time!";
    }
    if (Arg.sort) {
        ASSERT_FALSE(Arg.shuffle)
            << " cannot shuffle and sort at the same time!";
    }

    // Select device
    cuda_query(Arg.device_id);

    // Load mesh
    std::vector<std::vector<dataT>>    Verts;
    std::vector<std::vector<uint32_t>> Faces;

    if (!import_obj(Arg.obj_file_name, Verts, Faces)) {
        exit(EXIT_FAILURE);
    }

    if (Arg.shuffle) {
        shuffle_obj(Faces, Verts);
    }

    // Create RXMeshStatic instance. If Arg.sort is true, Faces and Verts will
    // be sorted based on the patching happening inside RXMesh
    RXMeshStatic rxmesh_static(Faces, Verts, Arg.sort, false);

    //*** RXMesh Impl
    mass_spring_rxmesh(rxmesh_static, Verts, Faces);
}

int main(int argc, char** argv)
{
    using namespace RXMESH;
    Log::init();

    ::testing::InitGoogleTest(&argc, argv);
    Arg.argv = argv;
    Arg.argc = argc;

    if (argc > 1) {
        if (cmd_option_exists(argv, argc + argv, "-h")) {
            // clang-format off
            RXMESH_INFO("\nUsage: MassSpring.exe < -option X>\n"
                        " -h:          Display this massage and exits\n"
                        " -input:      Input file. Input file should under the input/ subdirectory\n"
                        "              Default is {} \n"
                        "              Hint: Only accepts OBJ files\n"
                        " -o:          JSON file output folder. Default is {} \n"
                        " -num_run:    Number of iterations for performance testing. Default is {} \n"                        
                        " -s:          Shuffle input. Default is false.\n"
                        " -p:          Sort input using patching output. Default is false.\n"
                        " -device_id:  GPU device ID. Default is {}",
            Arg.obj_file_name, Arg.output_folder, Arg.num_run, Arg.device_id);
            // clang-format on
            exit(EXIT_SUCCESS);
        }

        if (cmd_option_exists(argv, argc + argv, "-num_run")) {
            Arg.num_run = atoi(get_cmd_option(argv, argv + argc, "-num_run"));
        }

        if (cmd_option_exists(argv, argc + argv, "-input")) {
            Arg.obj_file_name =
                std::string(get_cmd_option(argv, argv + argc, "-input"));
        }
        if (cmd_option_exists(argv, argc + argv, "-o")) {
            Arg.output_folder =
                std::string(get_cmd_option(argv, argv + argc, "-o"));
        }
        if (cmd_option_exists(argv, argc + argv, "-device_id")) {
            Arg.device_id =
                atoi(get_cmd_option(argv, argv + argc, "-device_id"));
        }
        if (cmd_option_exists(argv, argc + argv, "-s")) {
            Arg.shuffle = true;
        }
        if (cmd_option_exists(argv, argc + argv, "-ev")) {
            Arg.ev = true;
        }
        if (cmd_option_exists(argv, argc + argv, "-vv")) {
            Arg.vv = true;
        }
    }

    RXMESH_TRACE("input= {}", Arg.obj_file_name);
    RXMESH_TRACE("output_folder= {}", Arg.output_folder);
    RXMESH_TRACE("num_run= {}", Arg.num_run);
    RXMESH_TRACE("device_id= {}", Arg.device_id);

    return RUN_ALL_TESTS();
}
